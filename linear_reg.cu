
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void computeSigma(float *d_sigma, float *d_x, float *d_y, float *d_theta) {
	// compute sigma for linear regression

	// initialize values
	int idx = threadIdx.x;
	float x = d_x[idx], y = d_y[idx];
	// printf("%d: x = %f y = %f\n", idx, x, y);
	
	// compute sigma using the formula given
	d_sigma[idx] = ((d_theta[0] + d_theta[1] * x) - y) * ((d_theta[0] + d_theta[1] * x) - y);

}

long timediff(clock_t t1, clock_t t2) {
	long elapsed;
	elapsed = ((double)t2 - t1) / CLOCKS_PER_SEC * 1000;
	return elapsed;
}

int main() {
	// load input data on the host
	FILE *fp;
	fp = fopen("ex1data1.txt", "r");
	if (fp == NULL) {
		printf("Couldn't open file\n");
		return 1;
	}

	float h_x[97], h_y[97];

	for (int i = 0; i < 97; i++) {
		fscanf(fp, "%f,%f", &h_x[i], &h_y[i]);
	}

	fclose(fp);

	// generate other data on the host
	int h_size = sizeof(h_y) / sizeof(float);
	float h_sigma[h_size];
	float h_theta[2] = {0, 0};
	clock_t start, end;

	// declare GPU memory pointers
	float * d_sigma, * d_x, * d_y, * d_theta;

	// allocate GPU memory
	hipMalloc((void **) &d_sigma, sizeof(h_sigma));
	hipMalloc((void **) &d_x, sizeof(h_x));
	hipMalloc((void **) &d_y, sizeof(h_y));
	hipMalloc((void **) &d_theta, sizeof(h_theta));

	// transfer the data to the GPU
	hipMemcpy(d_sigma, h_sigma, sizeof(h_sigma), hipMemcpyHostToDevice);
	hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice);
	hipMemcpy(d_theta, h_theta, sizeof(h_theta), hipMemcpyHostToDevice);
	
	start = clock();
	// launch the kernel
	computeSigma<<<1, h_size>>>(d_sigma, d_x, d_y, d_theta);

	// copy back the result to the CPU
	hipMemcpy(h_sigma, d_sigma, sizeof(h_sigma), hipMemcpyDeviceToHost);

	// add each value in sigma to compute cost
	float J = 0;
	for (int j = 0; j < h_size; j++) {
		J += h_sigma[j];
	}
	J = J / (2 * h_size);
	end = clock();	
	
	// print out the result
	printf("Initial cost = %f Elapsed time is %lu ms\n", J, (end - start));

	return 0;
}
